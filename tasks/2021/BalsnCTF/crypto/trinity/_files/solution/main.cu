#include "hip/hip_runtime.h"
// nvcc -O3 main.cu && time ./a.out
#include <cstdio>
#include <cstdint>
#include <cassert>
#include <hip/hip_runtime.h>
#include <unordered_map>
#include <map>
#include <algorithm>
#include <vector>
#include <sys/time.h>

#include <thrust/sequence.h>
#include <thrust/execution_policy.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/fill.h>


using namespace std;

// Config

// GPU config
// #thread per block = warp size
#define THREAD_NUM 32
// #step per kernel invocation
#define LOOP_COUNT 1024

// Search config
// Search for 2^MAX_STEP_B steps (45 tooks at most 30min on T4)
// #define MAX_STEP_B 45
#define MAX_STEP_B 45
// Save approx 2^MAX_TRACE_B traces (26 tooks about 5GB)
// #define MAX_TRACE_B 26
#define MAX_TRACE_B 26
// Split to MAX_ROUND and search for collisions in the available traces after each round
#define MAX_ROUND 10

#define HASHBITS 64

#define MAX_STEP (1ULL<<MAX_STEP_B)
#define SHIFT (31 - (MAX_STEP_B - MAX_TRACE_B))
#define is_dis(x) (! (((uint32_t)(x)<<SHIFT)) )
#define HSHIFT (64 - HASHBITS)

static_assert(SHIFT >= 0);
static_assert(HSHIFT >= 0);


// Helpers

#define likely(x)       __builtin_expect((x),1)
#define unlikely(x)     __builtin_expect((x),0)

#define chkCuda(code) { _chkCuda((code), __FILE__, __LINE__); }
static inline void _chkCuda(hipError_t code, const char *file, int line) {
   if (code != hipSuccess) {
      fprintf(stderr,"\nCUDA Runtime Error: %s @ %s:%d\n", hipGetErrorString(code), file, line);
      exit(code);
   }
}

#define FUNC __host__ __device__ static inline
#ifdef __CUDA_ARCH__
#define CONSTANT __constant__ const
#else
#define CONSTANT const
#endif

FUNC uint64_t xrand(uint64_t x) {
    x ^= x >> 12; // a
    x ^= x << 25; // b
    x ^= x >> 27; // c
    return x * 0x2545f4914f6cdd1dULL;
}


// Function config
typedef uint32_t state_t[16];

const uint32_t CHUNK_START         = 1 << 0;
const uint32_t CHUNK_END           = 1 << 1;
const uint32_t ROOT                = 1 << 3;
const uint32_t KEYED_HASH          = 1 << 4;

CONSTANT uint32_t IV[8] = {
    0x6A09E667, 0xBB67AE85, 0x3C6EF372, 0xA54FF53A, 0x510E527F, 0x9B05688C, 0x1F83D9AB, 0x5BE0CD19,
};

CONSTANT uint32_t MSG_PERMUTATION[16] = {2, 6, 3, 10, 7, 0, 4, 13, 1, 11, 12, 5, 9, 14, 15, 8};

// Rotate right
FUNC uint32_t blake3_ror (const uint32_t x, const uint32_t s) {
    return (x >> s) | (x << (32 - s));
}

// The mixing function, G, which mixes either a column or a diagonal.
FUNC void blake3_g(state_t& state, uint32_t a, uint32_t b, uint32_t c, uint32_t d, uint32_t mx, uint32_t my) {
    state[a] = state[a] + state[b] + mx;
    state[d] = blake3_ror(state[d] ^ state[a], 16);
    state[c] = state[c] + state[d];
    state[b] = blake3_ror(state[b] ^ state[c], 12);
    state[a] = state[a] + state[b] + my;
    state[d] = blake3_ror(state[d] ^ state[a], 8);
    state[c] = state[c] + state[d];
    state[b] = blake3_ror(state[b] ^ state[c], 7);
}

FUNC void blake3_round(state_t& state, state_t& m) {
    // Mix the columns.
    blake3_g(state, 0, 4, 8, 12, m[0], m[1]);
    blake3_g(state, 1, 5, 9, 13, m[2], m[3]);
    blake3_g(state, 2, 6, 10, 14, m[4], m[5]);
    blake3_g(state, 3, 7, 11, 15, m[6], m[7]);
    // Mix the diagonals.
    blake3_g(state, 0, 5, 10, 15, m[8], m[9]);
    blake3_g(state, 1, 6, 11, 12, m[10], m[11]);
    blake3_g(state, 2, 7, 8, 13, m[12], m[13]);
    blake3_g(state, 3, 4, 9, 14, m[14], m[15]);
}

FUNC void blake3_permute(state_t& m) {
    state_t tmp;
    for (int i=0; i<16; i++) {
        tmp[i] = m[MSG_PERMUTATION[i]];
    }
    memcpy(m, tmp, sizeof(tmp));
}

FUNC void blake3(uint32_t *key, state_t& block, uint32_t block_len) {
    uint32_t flags = KEYED_HASH | CHUNK_START | CHUNK_END | ROOT;
    state_t state = {
        key[0], key[1], key[2], key[3], key[4], key[5], key[6], key[7], 
        IV[0], IV[1], IV[2], IV[3], 0, 0, block_len, flags,
    };

    blake3_round(state, block); // round 1
    blake3_permute(block);
    blake3_round(state, block); // round 2
    blake3_permute(block);
    blake3_round(state, block); // round 3
    blake3_permute(block);
    blake3_round(state, block); // round 4
    blake3_permute(block);
    blake3_round(state, block); // round 5
    blake3_permute(block);
    blake3_round(state, block); // round 6
    blake3_permute(block);
    blake3_round(state, block); // round 7

    for (int i=0; i<8; i++) {
        block[i] = state[i] ^ state[i + 8];
        block[i+8] = state[i + 8] ^ key[i];
    }
}

FUNC uint64_t step(uint64_t k) {
    uint32_t key[8] = { 1768647031, 1768187248, 1919888993, 1769418599, 1412393323, 1768843634, 538999156, 538976288 };

    state_t block = { (uint32_t)k, (uint32_t)(k>>32), 0 };
    blake3((uint32_t*)key, block, 8);
    return ((((uint64_t)block[1] << 32) | block[0]) << HSHIFT) >> HSHIFT;
}


// Main algo

struct trace_t {
    uint64_t val;
    uint64_t end;
    uint64_t len;
};

struct cut_t {
    uint64_t val;
    uint64_t len;
};

__device__ int _count;

__global__ void init (uint64_t seed, trace_t* A, uint64_t n) {
    uint64_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= n) return;
    auto&& X = A[tid];

    _count = 0;
    uint64_t x = xrand(seed + xrand(tid) + xrand(clock64()));
    X = {.val=x, .end=x, .len=0};
}

__global__ void gen_traces (trace_t* A, trace_t* O, uint64_t n) {
    uint64_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= n) return;
    auto&& X = A[tid];

    uint64_t x = X.end, l = X.len;

    for (int i=0; i<LOOP_COUNT; i++) {
        // Step x
        x = step(x); l++;

        // Break the loop works much faster than start a new trace on pre-Volta GPU. (2x)
        // Search for distinguished point
        if (unlikely(is_dis(x))) { break; }
    }

    __syncwarp();

    X.end = x;
    X.len = l;

    if (unlikely(is_dis(x))) {
        // output result
        int idx = atomicAdd(&_count, 1);

        // check overflow
        if (likely(idx < n)) {
            O[idx] = X;
        }

        // new trace
        x = xrand(x + xrand(tid) + xrand(clock64()));
        X = {.val=x, .end=x, .len=0};
    }
}

__global__ void cut_traces (cut_t* C, uint64_t n) {
    uint64_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= n) return;
    auto&& X = C[tid];

    uint64_t x = X.val, l = X.len;

    while (l--) { x = step(x); }

    __syncwarp();

    X.val = x;
}

uint64_t timer(bool reset=false) {
    static struct timeval tv_start, tv_end;

    if (tv_start.tv_sec == 0 || reset) {
        gettimeofday(&tv_start,NULL);
    }

    gettimeofday(&tv_end,NULL);
    uint64_t start = 1000000 * tv_start.tv_sec + tv_start.tv_usec;
    uint64_t end = 1000000 * tv_end.tv_sec + tv_end.tv_usec;
    return end - start;
}

struct tuple_hash : public unary_function<tuple<uint64_t, uint64_t>, size_t> {
    size_t operator()(const tuple<uint64_t, uint64_t>& k) const {
        return xrand(xrand(get<0>(k)) + xrand(get<1>(k)));
    }
};

unordered_map<uint64_t, map<uint64_t, uint64_t> > traces;
unordered_map<tuple<uint64_t, uint64_t>, uint64_t, tuple_hash> cut_cache;
uint64_t tcount = 0;
uint64_t steps = 0;
int done = 0;

void print_time(uint64_t x) {
    uint64_t s = x / 1e6;
    uint64_t m = s / 60;
    uint64_t h = m / 60;
    fprintf(stderr, "%02u:%02u:%02u", h, m%60, s%60);
}

void print_dur(double x, const char* suffix) {
    if (x < 1e3) {fprintf(stderr, "%8.2lf us%s", x, suffix); return;}
    if (x < 1e6) {fprintf(stderr, "%8.2lf ms%s", x / 1e3, suffix); return;}
    if (x < 6e7) {fprintf(stderr, "%5.2lf s%s", x / 1e6, suffix); return;}
    if (x < 36e8) {fprintf(stderr, "%5.2lf m%s", x / 6e7, suffix); return;}
    fprintf(stderr, "%5.2lf h%s", x / 36e8, suffix); return;
}

void stats(bool force=false) {
    static uint64_t last_dur = 0;
    uint64_t dur = timer();
    if (force || (dur >> 16) != (last_dur >> 16)) {
        double prog = (double) steps / MAX_STEP;
        uint64_t eta = (double)dur / prog;

        fprintf(stderr, "\r%6.2lf%% [", prog * 100);
        print_time(dur);
        fprintf(stderr, " / ");
        print_time(eta);
        fprintf(stderr, "] %.2lfG steps, %llu traces, %llu groups, ", 
            (double) steps / 1e9,
            tcount,
            traces.size()
        );
        print_dur((double)dur / tcount, "/T, ");
        print_dur((double)dur / traces.size(), "/G, ");
        fprintf(stderr, "%.2lf GS/s", (double)steps / dur / 1000);
        last_dur = dur;
    }
}

int main() {
    int ngpu = 0;
    hipGetDeviceCount(&ngpu);
    fprintf(stderr, "[+] GPU Count: %lld\n", ngpu);
    if(ngpu == 0) { exit(0); }

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    fprintf(stderr, "[+] SM Count: %llu\n", prop.multiProcessorCount);
    uint64_t batch = prop.multiProcessorCount * 128 * 4;
    fprintf(stderr, "[+] Batch size: %llu\n", batch);
    uint64_t block_num = (batch + THREAD_NUM - 1) / THREAD_NUM;
    fprintf(stderr, "[+] Blocks: %llu\n", block_num);

    trace_t *O = new trace_t[batch], *_O, *_A;
    cut_t *C = new cut_t[batch], *_C;

    chkCuda( hipMalloc(&_O, sizeof(_O[0]) * batch) );
    chkCuda( hipMalloc(&_A, sizeof(_A[0]) * batch) );
    chkCuda( hipMalloc(&_C, sizeof(_C[0]) * batch) );

    fprintf(stderr, "[*] Initializing\n");
    init<<<block_num, THREAD_NUM>>>(time(NULL), _A, batch);
    chkCuda( hipPeekAtLastError() );
    chkCuda( hipDeviceSynchronize() );

    fprintf(stderr, "[*] Start searching\n");

    timer();

    for (int round=0; !done && round<MAX_ROUND; round++) {
        while (steps < MAX_STEP * (round + 1) / MAX_ROUND) {
            gen_traces<<<block_num, THREAD_NUM>>>(_A, _O, batch);
            chkCuda( hipPeekAtLastError() );
            chkCuda( hipDeviceSynchronize() );

            // Retrieve results
            int count;
            chkCuda( hipMemcpyFromSymbol(&count, HIP_SYMBOL(_count), sizeof(count)) );
            assert( count <= batch );
            if (count) {
                chkCuda( hipMemcpy(O, _O, sizeof(trace_t) * count, hipMemcpyDeviceToHost) );

                tcount += count;

                // Save traces
                for (int i=0; i<count; i++) {
                    auto&& X = O[i];
                    steps += X.len;
                    traces[X.end][X.val] = X.len;
                }

                // Reset buffer
                count = 0;
                chkCuda( hipMemcpyToSymbol(HIP_SYMBOL(_count), &count, sizeof(count)) );
            }

            stats();
        }

        stats(true);
        fprintf(stderr, "\n");

        unordered_map<uint64_t, map<uint64_t, uint64_t> > groups;
        for (auto& it : traces) { if (it.second.size() > 2) groups[it.first] = it.second; }
        fprintf(stderr, "%16llu groups", groups.size());

        vector<trace_t> cached;
        vector<trace_t> cutv;
        vector<cut_t> cuts;
        thrust::host_vector<uint64_t> lens;
        while(!done && groups.size()) {
            done = true;

            // Split groups with too many collision
            uint64_t cut_steps = 0;
            for (auto& it : groups) {
                auto&& g = it.first;
                auto&& A = it.second;

                uint64_t maxlen = 0;
                for (auto& it : A) { maxlen = max(maxlen, it.second); }

                if (maxlen == 1) { continue; } // finished
                done = false;

                uint64_t cutpoint = 1ULL << (63 - __builtin_clzll(maxlen-1));
                assert(cutpoint < maxlen);
                assert(cutpoint >= maxlen / 2);

                map<uint64_t, uint64_t> B = move(A);
                assert(A.size() == 0);
                for (auto& it : B) {
                    uint64_t v = it.first;
                    uint64_t len = it.second;
                    if (len > cutpoint) { // cut
                        uint64_t s = len - cutpoint;
                        auto&& it = cut_cache.find({v, s});
                        if (it != cut_cache.end()) {
                            uint64_t m = it->second;
                            cached.push_back({.val=v, .end=m, .len=s});
                            A[m] = cutpoint;
                        } else {
                            cutv.push_back({.val=v, .end=g, .len=len});
                            cuts.push_back({.val=v, .len=s});
                            lens.push_back(s);
                            cut_steps += s;
                        }
                    } else { // keep
                        A[v] = len;
                    }
                }
            }
            if (done) break;
            for (auto&& X : cached) { groups[X.end][X.val] = X.len; }
            cached.clear();

            fprintf(stderr, ", %16d traces, %16llu steps.", cuts.size(), cut_steps);

            thrust::host_vector<uint64_t> idx;
            {
                thrust::device_vector<uint64_t> _lens = lens;
                thrust::device_vector<uint64_t> _idx(lens.size());
                thrust::sequence(_idx.begin(), _idx.end());
                thrust::sort_by_key(_lens.begin(), _lens.end(), _idx.begin());
                chkCuda( hipDeviceSynchronize() );
                idx = _idx;
            }
            lens.clear();
            fprintf(stderr, ".     ");

            for (uint64_t i=0; i<cuts.size(); i+=batch) {
                fprintf(stderr, "\b\b\b\b%3llu%%", i * 100 / cuts.size());
                uint64_t n = min((uint64_t)cuts.size() - i, (uint64_t)batch);
                for (int k=0; k<n; k++) { C[k] = cuts[idx[i+k]]; }
                chkCuda( hipMemcpy(_C, C, sizeof(cut_t) * n, hipMemcpyHostToDevice) );
                chkCuda( hipDeviceSynchronize() );

                cut_traces<<<block_num, THREAD_NUM>>>(_C, n);
                chkCuda( hipPeekAtLastError() );
                chkCuda( hipDeviceSynchronize() );

                chkCuda( hipMemcpy(C, _C, sizeof(cut_t) * n, hipMemcpyDeviceToHost) );
                chkCuda( hipDeviceSynchronize() );
                for (int k=0; k<n; k++) { cuts[idx[i+k]] = C[k]; }
            }
            fprintf(stderr, "\b\b\b\b\b.     \b\b\b\b\b");

            {
                for (int i=0; i<cuts.size(); i++) {
                    auto&& X = cutv[i];
                    auto&& C = cuts[i];
                    groups[C.val][X.val] = C.len;
                    groups[X.end][C.val] = X.len - C.len;
                    cut_cache[{X.val, C.len}] = C.val;
                }
            }
            cutv.clear();
            cuts.clear();

            fprintf(stderr, " done\n");
            steps += cut_steps;

            // Remove groups without enough collision
            vector<uint64_t> remove;
            for (auto& it : groups) { if (it.second.size() <= 2) remove.push_back(it.first); }
            for (auto g : remove) { groups.erase(g); }
            fprintf(stderr, "%16llu groups", groups.size());
        }
        fprintf(stderr, "\n");

        if (done && groups.size()) {
            for (auto& it : groups) {
                auto&& g = it.first;
                auto&& A = it.second;
                uint64_t maxlen = 0;
                for (auto& it : A) { maxlen = max(maxlen, it.second); }
                if (maxlen != 1) { continue ;}

                fprintf(stderr, "%016llx:\n", __builtin_bswap64(g));
                for (auto& it : A) {
                    uint64_t v = it.first;
                    fprintf(stderr, "  %016llx -> %016llx\n", __builtin_bswap64(v), __builtin_bswap64(step(v)));
                }
            }
            break;
        }
    }
}
